#include "hip/hip_runtime.h"
/*********************************************************************************************************************************
Service name:               Com_ReceiveShadowSignal
Service ID:                    0x0f
Parameters (in):           SignalId--> Id of group signal to be received.
						   SignalDataPtr --> Reference to the group signal data in which to store the received data.
Parameters (inout):            None
Parameters (out):              None
Return value:                  None
Description:        The service Com_ReceiveShadowSignal updates the group signal which is referenced by SignalDataPtr with the data in the shadow buffer.
*******************************************************************************************************************************/
__global__ void Com_Recieve_shadow_signal(Com_SignalIdType* SignalId_arr, void* SignalDataPtr_arr, uint8* Buffer)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < ComMaxGroupSignalCnt)
	{
		if (SignalId_arr[id] < ComMaxGroupSignalCnt)
		{
			((uint8*)SignalDataPtr_arr)[id] = Buffer[SignalId_arr[id]];
		}
	}
}
void Com_ReceiveShadowSignal_GPU(Com_SignalIdType* SignalId_arr, void* SignalDataPtr_arr)
{
	Com_SignalIdType* SignalId_GPU_arr;

	uint8* SignalDataPtr_GPU_arr, * Buffer_GPU;
	uint8 Buffer[ComMaxGroupSignalCnt];
	hipEvent_t start, stop;
	hipMalloc((void**)&SignalId_GPU_arr, ComMaxGroupSignalCnt * sizeof(Com_SignalIdType));
	hipMalloc((void**)&SignalDataPtr_GPU_arr, ComMaxGroupSignalCnt * sizeof(uint8));
	hipMalloc((void**)&Buffer_GPU, ComMaxGroupSignalCnt * sizeof(uint8));

	for (int i = 0; i < ComMaxGroupSignalCnt; i++)
	{
		Buffer[i] = *(Com.ComConfig.ComGroupSignal[i].ComBufferRef);
	}

	hipEvent_t start1, stop1;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventRecord(start1);
	hipMemcpy(SignalId_GPU_arr, SignalId_arr, ComMaxGroupSignalCnt * sizeof(Com_SignalIdType), hipMemcpyHostToDevice);
	hipMemcpy(Buffer_GPU, Buffer, ComMaxGroupSignalCnt * sizeof(uint8), hipMemcpyHostToDevice);
	hipEventRecord(stop1);
	hipEventSynchronize(stop1);
	hipEventElapsedTime(&gpu_time[0], start1, stop1);
	hipEvent_t start2, stop2;
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	hipEventRecord(start2);
	Com_Recieve_shadow_signal << <THREADS_PER_BLOCK, BLOCKS >> > (SignalId_GPU_arr, SignalDataPtr_GPU_arr, Buffer_GPU);
	hipDeviceSynchronize();
	hipEventRecord(stop2);
	hipEventSynchronize(stop2);
	hipEventElapsedTime(&gpu_time[1], start2, stop2);
	hipEvent_t start3, stop3;
	hipEventCreate(&start3);
	hipEventCreate(&stop3);
	hipEventRecord(start3);
	hipMemcpy(SignalDataPtr_arr, SignalDataPtr_GPU_arr, ComMaxGroupSignalCnt * sizeof(uint8), hipMemcpyDeviceToHost);
	hipEventRecord(stop3);
	hipEventSynchronize(stop3);
	hipEventElapsedTime(&gpu_time[2], start3, stop3);
}
void Com_ReceiveShadowSignal(Com_SignalIdType SignalId, void* SignalDataPtr)
{
	uint8 ComGroupSignalIndex;

	/* Check that the group signal ID is a valid ID*/
	if (SignalId < ComMaxGroupSignalCnt)
	{
		/*Find GroupSignal with such ID*/
		for (ComGroupSignalIndex = 0; ComGroupSignalIndex < ComMaxGroupSignalCnt; ComGroupSignalIndex++)
		{
			if (Com.ComConfig.ComGroupSignal[ComGroupSignalIndex].ComHandleId == SignalId)
			{
				memcpy(SignalDataPtr, Com.ComConfig.ComGroupSignal[ComGroupSignalIndex].ComBufferRef, Com.ComConfig.ComGroupSignal[ComGroupSignalIndex].ComSignalLength);

				return;
			}
			else
			{

			}
		}
	}
	else
	{

	}
}
