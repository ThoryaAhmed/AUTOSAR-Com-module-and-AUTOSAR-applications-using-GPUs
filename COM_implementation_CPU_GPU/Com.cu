#include "hip/hip_runtime.h"
/*********************************************************************************************************************************
Service name:               Com_ReceiveShadowSignal
Service ID:                    0x0f
Parameters (in):           SignalId--> Id of group signal to be received.
						   SignalDataPtr --> Reference to the group signal data in which to store the received data.
Parameters (inout):            None
Parameters (out):              None
Return value:                  None
Description:        The service Com_ReceiveShadowSignal updates the group signal which is referenced by SignalDataPtr with the data in the shadow buffer.
*******************************************************************************************************************************/
__global__ void Com_Recieve_shadow_signal(Com_SignalIdType* SignalId_arr, void* SignalDataPtr_arr, uint8* Buffer)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < ComMaxGroupSignalCnt)
	{
		if (SignalId_arr[id] < ComMaxGroupSignalCnt)
		{
			((uint8*)SignalDataPtr_arr)[id] = Buffer[SignalId_arr[id]];
		}
	}
}
void Com_ReceiveShadowSignal_GPU(Com_SignalIdType* SignalId_arr, void* SignalDataPtr_arr)
{
	Com_SignalIdType* SignalId_GPU_arr;

	uint8* SignalDataPtr_GPU_arr, * Buffer_GPU;
	uint8 Buffer[ComMaxGroupSignalCnt];
	hipEvent_t start, stop;
	hipMalloc((void**)&SignalId_GPU_arr, ComMaxGroupSignalCnt * sizeof(Com_SignalIdType));
	hipMalloc((void**)&SignalDataPtr_GPU_arr, ComMaxGroupSignalCnt * sizeof(uint8));
	hipMalloc((void**)&Buffer_GPU, ComMaxGroupSignalCnt * sizeof(uint8));

	for (int i = 0; i < ComMaxGroupSignalCnt; i++)
	{
		Buffer[i] = *(Com.ComConfig.ComGroupSignal[i].ComBufferRef);
	}

	hipEvent_t start1, stop1;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventRecord(start1);
	hipMemcpy(SignalId_GPU_arr, SignalId_arr, ComMaxGroupSignalCnt * sizeof(Com_SignalIdType), hipMemcpyHostToDevice);
	hipMemcpy(Buffer_GPU, Buffer, ComMaxGroupSignalCnt * sizeof(uint8), hipMemcpyHostToDevice);
	hipEventRecord(stop1);
	hipEventSynchronize(stop1);
	hipEventElapsedTime(&gpu_time[0], start1, stop1);
	hipEvent_t start2, stop2;
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	hipEventRecord(start2);
	Com_Recieve_shadow_signal << <THREADS_PER_BLOCK, BLOCKS >> > (SignalId_GPU_arr, SignalDataPtr_GPU_arr, Buffer_GPU);
	hipDeviceSynchronize();
	hipEventRecord(stop2);
	hipEventSynchronize(stop2);
	hipEventElapsedTime(&gpu_time[1], start2, stop2);
	hipEvent_t start3, stop3;
	hipEventCreate(&start3);
	hipEventCreate(&stop3);
	hipEventRecord(start3);
	hipMemcpy(SignalDataPtr_arr, SignalDataPtr_GPU_arr, ComMaxGroupSignalCnt * sizeof(uint8), hipMemcpyDeviceToHost);
	hipEventRecord(stop3);
	hipEventSynchronize(stop3);
	hipEventElapsedTime(&gpu_time[2], start3, stop3);
}
void Com_ReceiveShadowSignal(Com_SignalIdType SignalId, void* SignalDataPtr)
{
	uint8 ComGroupSignalIndex;

	/* Check that the group signal ID is a valid ID*/
	if (SignalId < ComMaxGroupSignalCnt)
	{
		/*Find GroupSignal with such ID*/
		for (ComGroupSignalIndex = 0; ComGroupSignalIndex < ComMaxGroupSignalCnt; ComGroupSignalIndex++)
		{
			if (Com.ComConfig.ComGroupSignal[ComGroupSignalIndex].ComHandleId == SignalId)
			{
				memcpy(SignalDataPtr, Com.ComConfig.ComGroupSignal[ComGroupSignalIndex].ComBufferRef, Com.ComConfig.ComGroupSignal[ComGroupSignalIndex].ComSignalLength);

				return;
			}
			else
			{

			}
		}
	}
	else
	{

	}
}

uint8 Com_ReceiveSignalGroup_GPU(Com_SignalGroupIdType SignalGroupId)
{
	/* Definition of Variables */
	Com_GroupSignalType* ComGroupSignalLocal = NULL;
	Com_IPduType* ComIPduLocal = NULL;

	/***************************/

	/***************************************************************************************************************************/
	/*          The service Com_ReceiveSignalGroup shall copy the received singal group from I-PDU to shadow buffer            */
	/* After this call, the group signals could be copied from the shadow buffer to the rte by calling Com_ReceiveShadowSignal */
	/***************************************************************************************************************************/

	/* Check that the id is valid */
	if (SignalGroupId <= ComMaxSignalGroupCnt)
	{
		/* Get the signal */
		const Com_SignalGroupType* ComSignalGroup = &Com.ComConfig.ComSignalGroup[SignalGroupId];

		/* Get IPDU */
		ComIPduLocal = &Com.ComConfig.ComIPdu[ComSignalGroup->ComIPduHandleId];

		uint8 ComSignalGroupIndex = ComSignalGroup->ComIPduHandleIndex;

		/* Initialize GPU Streams */
		int N_STREAMS = ComIPduLocal->ComIPduSignalGroupRef[ComSignalGroupIndex]->ComGroupSignalsNumbers;
		hipStream_t* stream = new hipStream_t[N_STREAMS];

		for (uint8 ComGroupSignalIndex = 0; ComGroupSignalIndex < N_STREAMS; ComGroupSignalIndex++)
		{
			/* Create Stream */
			hipStreamCreate(&stream[ComGroupSignalIndex]);

			/*Get Group Signal*/
			ComGroupSignalLocal = ComIPduLocal->ComIPduSignalGroupRef[ComSignalGroupIndex]->ComGroupSignalRef[ComGroupSignalIndex];

			/* GPU Operations Begin here */
			uint8 threads = NUM_THREADS;
			uint16 blocks = ceil(ComGroupSignalLocal->ComBitSize / (float)threads) * Signals_Factor;
			uint8 length = ComIPduLocal->ComIPduLength;

			uint8* h_output_values = ComGroupSignalLocal->ComBufferRef;
			uint8* h_input_values = ComIPduLocal->ComBufferRef;
			uint8* h_data = h_input_values;
			uint16 h_size_out = ComGroupSignalLocal->ComBitSize;
			uint8 h_size_in = ComIPduLocal->ComIPduLength * 8;
			uint8 h_bit_position = ComGroupSignalLocal->ComBitPosition;

			uint8 h_length_in = ComIPduLocal->ComIPduLength;
			uint8 h_length_out = ComGroupSignalLocal->ComSignalLength;
			
			// declare GPU memory pointers
			uint8* d_output_values, * d_input_values , *d_data;
			uint16* d_size_out, * d_size_in;
			uint32* d_bit_position;

			hipEvent_t start3, stop3;
			hipEventCreate(&start3);
			hipEventCreate(&stop3);
			hipEventRecord(start3);

			// allocate GPU memory
			hipMalloc((void**)&d_output_values, h_length_out * sizeof(uint8));
			hipMalloc((void**)&d_input_values, h_length_in * sizeof(uint8));
			hipMalloc((void**)&d_data, Signals_Factor* h_length_in * sizeof(uint8));

			// transfer the input array to the GPU
			hipMemcpyAsync(d_output_values, h_output_values, h_length_out * sizeof(uint8), hipMemcpyHostToDevice);
			hipMemcpyAsync(d_input_values, h_input_values, h_length_in * sizeof(uint8), hipMemcpyHostToDevice);
			if (Signals_Factor < 1000) {
				hipMemcpyAsync(d_data, h_input_values, Signals_Factor * h_length_in, hipMemcpyHostToDevice);
				hipMemcpyAsync(d_data, h_input_values, Signals_Factor * h_length_out, hipMemcpyHostToDevice);
			}
			else {
				hipMemcpyAsync(d_data, h_input_values, Signals_Factor/2 * h_length_in, hipMemcpyHostToDevice);
				hipMemcpyAsync(d_data, h_input_values, Signals_Factor/2 * h_length_out, hipMemcpyHostToDevice);
				hipMemcpyAsync(d_data, h_input_values, Signals_Factor/2 * h_length_out, hipMemcpyHostToDevice);
				hipMemcpyAsync(d_data, h_input_values, Signals_Factor/2 * h_length_out, hipMemcpyHostToDevice);
			}

			hipEventRecord(stop3);
			hipEventSynchronize(stop3);


			hipEventElapsedTime(&gpu_time[0], start3, stop3);

			hipEvent_t start2, stop2;
			hipEventCreate(&start2);
			hipEventCreate(&stop2);
			hipEventRecord(start2);
			///////////////////////////////////////////////* Output Values *//* Input Values *//*size o/p*//*size of i/p*//* bit position of The signal */
			Unpacking_Bits_kernel << < blocks, threads, threads * sizeof(uint8), stream[ComGroupSignalIndex] >> > (d_output_values, d_input_values, h_size_out, h_size_in, h_bit_position);
			hipDeviceSynchronize();

			hipEventRecord(stop2);
			hipEventSynchronize(stop2);


			hipEventElapsedTime(&gpu_time[1], start2, stop2);
			hipEventDestroy(start2);
			hipEventDestroy(stop2);
			// Return the results to the signal

			hipEvent_t start4, stop4;
			hipEventCreate(&start4);
			hipEventCreate(&stop4);
			hipEventRecord(start4);

			// transfer the input array to the GPU
			hipMemcpyAsync(h_output_values, d_output_values,  h_length_out * sizeof(uint8), hipMemcpyDeviceToHost);
			if (Signals_Factor < 1000) {
				hipMemcpyAsync(h_data, d_data, Signals_Factor * h_length_out * sizeof(uint8), hipMemcpyDeviceToHost);
			}
			else {
				hipMemcpyAsync(h_data, d_data, Signals_Factor/2 * h_length_out * sizeof(uint8), hipMemcpyDeviceToHost);
				hipMemcpyAsync(h_data, d_data, Signals_Factor/2 * h_length_out * sizeof(uint8), hipMemcpyDeviceToHost);

			}
			hipEventRecord(stop4);
			hipEventSynchronize(stop4);

			hipEventElapsedTime(&gpu_time[2], start4, stop4);

			hipFree(d_data);
		}


		//}

		//hipDeviceReset();
		free(stream);
		return E_OK;

	}
	else
	{
	}

	return COM_SERVICE_NOT_AVAILABLE;
}

/*********************************************************************************************************************************
 Service name:               Com_UpdateShadowSignal_GPU
 Service ID:                    0x0c
 Parameters (in):           SignalId--> Id of group signal to be updated.(unit16)
							SignalDataPtr --> Reference to the group signal data to be updated.
 Parameters (inout):            None
 Parameters (out):              None
 Return value:                  None
 Description:        The service Com_UpdateShadowSignal updates a group signal with the data referenced by SignalDataPtr.
 *******************************************************************************************************************************/
__global__ void copy_data_to_shadowBuffer_on_Kernel(uint8* d_signalgroup_in, uint8 SignalDataPtr, uint8 ID)
{
	uint8 ThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;

	/* Check that the group signal ID is a valid ID*/

	if (ThreadIndex < ComMaxGroupSignalCnt && ThreadIndex == ID)
	{
		/* Copy group signal to group signal buffer */
		d_signalgroup_in[0] = SignalDataPtr;
	}


}
void Com_UpdateShadowSignal_GPU(Com_SignalIdType SignalId, const void* SignalDataPtr)
{
	uint8 SignalData = *((uint8*)SignalDataPtr);
	Com_GroupSignalType ComGroupSignalLocal = Com.ComConfig.ComGroupSignal[SignalId];
	/* Get the group_signal ID */
	uint8 ID = Com.ComConfig.ComGroupSignal[SignalId].ComHandleId;

	uint8* h_data = 0;

	uint8 h_Length = ComGroupSignalLocal.ComSignalLength;
	/*GPU Variables*/
	uint8* d_GroupSignalLocal;
	uint8* d_data;

	hipEvent_t start1, stop1;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventRecord(start1);

	/*allocate GPU memory*/
	hipMalloc((void**)&d_GroupSignalLocal, h_Length * sizeof(uint8));
	hipMalloc((void**)&d_data, Signals_Factor * h_Length * sizeof(uint8));

	hipEventRecord(stop1);
	hipEventSynchronize(stop1);

	hipEventElapsedTime(&gpu_time[0], start1, stop1);

	hipEvent_t start2, stop2;
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	hipEventRecord(start2);

	/*Call Kernel*/
	copy_data_to_shadowBuffer_on_Kernel <<<1, ComMaxGroupSignalCnt * Signals_Factor>>> (d_GroupSignalLocal, SignalData, SignalId);
	hipDeviceSynchronize();

	hipEventRecord(stop2);
	hipEventSynchronize(stop2);
	hipEventElapsedTime(&gpu_time[0], start2, stop2);

	hipEvent_t start3, stop3;
	hipEventCreate(&start3);
	hipEventCreate(&stop3);
	hipEventRecord(start3);

	hipMemcpy(ComGroupSignalLocal.ComBufferRef, d_GroupSignalLocal, h_Length * sizeof(uint8), hipMemcpyDeviceToHost);

	hipEventRecord(stop3);
	hipEventSynchronize(stop3);
	hipEventElapsedTime(&gpu_time[1], start3, stop3);




	// Return the results to the signal

}

/*********************************************************************************************************************************
 Service name:               Com_UpdateShadowSignal_GPU_ARRAY
 Service ID:                    0x0c
 Parameters (in):           SignalId--> Id of group signal to be updated.(unit16)
							SignalDataPtr --> Reference to the group signal data to be updated.
 Parameters (inout):            None
 Parameters (out):              None
 Return value:                  None
 Description:        The service Com_UpdateShadowSignal updates a group signal with the data referenced by SignalDataPtr.
 *******************************************************************************************************************************/
__global__ void copy_data_to_shadowBuffer_on_Kernel_ARRAY(uint8* d_signalgroup_in, uint8* SignalDataPtr)
{
	uint8 ThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;

	/* Check that the group signal ID is a valid ID*/

	if (ThreadIndex < ComMaxGroupSignalCnt)
	{
		/* Copy group signal to group signal buffer */
		d_signalgroup_in[ThreadIndex] = SignalDataPtr[ThreadIndex];
	}


}

void Com_UpdateShadowSignal_GPU_ARRAY(uint8* SignalIds, uint8* SignalDataPtr)
{
	Com_GroupSignalType ComGroupSignalLocal;
	/*allocate CPU memory*/
	uint8* c = (uint8*)malloc(ComMaxGroupSignalCnt * sizeof(uint8));

	/*GPU Variables*/
	uint8* d_GroupSignalLocal;
	uint8* d_SignalDataPtr;

	hipEvent_t start1, stop1;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventRecord(start1);

	/*allocate GPU memory*/
	hipMalloc((void**)&d_GroupSignalLocal, ComMaxGroupSignalCnt * sizeof(uint8));
	hipMalloc((void**)&d_SignalDataPtr, ComMaxGroupSignalCnt * sizeof(uint8));

	hipMemcpy(d_SignalDataPtr, SignalDataPtr, ComMaxGroupSignalCnt * sizeof(uint8), hipMemcpyHostToDevice);
	hipMemcpy(d_GroupSignalLocal, c, ComMaxGroupSignalCnt * sizeof(uint8), hipMemcpyHostToDevice);
	
	hipEventRecord(stop1);
	hipEventSynchronize(stop1);

	hipEventElapsedTime(&gpu_time[0], start1, stop1);

	hipEvent_t start2, stop2;
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	hipEventRecord(start2);

	/*Call Kernel*/
	copy_data_to_shadowBuffer_on_Kernel_ARRAY <<< 1, ComMaxGroupSignalCnt >>> (d_GroupSignalLocal, d_SignalDataPtr);

	hipDeviceSynchronize();

	hipEventRecord(stop2);
	hipEventSynchronize(stop2);

	hipEventElapsedTime(&gpu_time[1], start2, stop2);

	hipEvent_t start3, stop3;
	hipEventCreate(&start3);
	hipEventCreate(&stop3);
	hipEventRecord(start3);

	// Return the results to the signal
	hipMemcpy(c, d_GroupSignalLocal, ComMaxGroupSignalCnt * sizeof(uint8), hipMemcpyDeviceToHost);

	hipEventRecord(stop3);
	hipEventSynchronize(stop3);
	
	hipEventElapsedTime(&gpu_time[2], start3, stop3);


	int len = sizeof(c)/sizeof(uint8);

	for (uint8 i = 0; i < 5; i++)
	{
		*Com.ComConfig.ComGroupSignal[SignalIds[i]].ComBufferRef = c[i];
	}

}
